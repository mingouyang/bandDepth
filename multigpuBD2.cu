#include "hip/hip_runtime.h"
#include <R.h>
#include <stdint.h>

#define min(A,B)  ((A)<(B) ? (A) : (B))
#define max(A,B)  ((A)>(B) ? (A) : (B))
#define Data(i,j) data[(j) * (*row) + (i)] //R uses column-major order
#define Func(i,j) func[(i) * m + (j)]

#define numThread   256
#define MaxN      65536
#define MaxCol      256

static unsigned *count, *tmpCount, *f1, *f2, **gpuCount, **gpuF1, **gpuF2;
static float *func, **gpuFunc;

__global__
void kernel(float *func, unsigned *count, unsigned n, unsigned m,
	    unsigned *f1, unsigned *f2){
  __shared__ float minVector[MaxCol];
  __shared__ float maxVector[MaxCol];
  unsigned myFunc, i, j;
  float funcValue;

  if (threadIdx.x < m){
    funcValue = Func(f1[blockIdx.x], threadIdx.x); //func 1
    minVector[threadIdx.x] = funcValue;
    maxVector[threadIdx.x] = funcValue;
    funcValue = Func(f2[blockIdx.x], threadIdx.x); //func 2
    minVector[threadIdx.x] = min(minVector[threadIdx.x], funcValue);
    maxVector[threadIdx.x] = max(maxVector[threadIdx.x], funcValue);
  }
  __syncthreads();

  for (i=0; i<n; i += blockDim.x){
    myFunc = i + threadIdx.x;
    if (myFunc < n){
      for (j=0; j<m; j++){
	funcValue = Func(myFunc, j);
	if (funcValue < minVector[j] || funcValue > maxVector[j])
	  break;
      }
      if (j == m)
	atomicAdd(count + myFunc, 1);
    }
  }
}

extern "C"
void multigpuBD2(int *row, int *col, double *data, double *depth){
  unsigned n, m, chunk, size;
  uint64_t i, j, k, numPairs;
  int numGPU;

  n = *row;
  m = *col;
  hipGetDeviceCount(&numGPU);
  if (n > MaxN){
    fprintf(stderr, "number of rows cannot be more than %u\n", MaxN);
    exit(1);
  }
  if (m > MaxCol){
    fprintf(stderr, "number of columns cannot be more than %u\n", MaxCol);
    exit(1);
  }
  if (numGPU < 2){
    fprintf(stderr, "need more than 1 GPU\n");
    exit(1);
  }

  count    = (unsigned*)malloc(sizeof(unsigned) * n);
  tmpCount = (unsigned*)malloc(sizeof(unsigned) * n);
  func     =    (float*)malloc(sizeof(float) * n * m);
  for (i=0; i<n; i++){
    count[i] = 0;
    for (j=0; j<m; j++)
      Func(i, j) = Data(i, j);
    //data: column major, double
    //func: row major, float
  }
  numPairs = (uint64_t)n * (n-1) / 2;
  f1 = (unsigned*)malloc(sizeof(unsigned) * numPairs);
  f2 = (unsigned*)malloc(sizeof(unsigned) * numPairs);
  for (i=0, k=0; i<n; i++)
    for (j=i+1; j<n; j++)
      f1[k] = i, f2[k++] = j;

  chunk = (numPairs + numGPU - 1) / numGPU;
  gpuCount = (unsigned**)malloc(numGPU * sizeof(unsigned*));
  gpuF1    = (unsigned**)malloc(numGPU * sizeof(unsigned*));
  gpuF2    = (unsigned**)malloc(numGPU * sizeof(unsigned*));
  gpuFunc  =    (float**)malloc(numGPU * sizeof(float*));
  for (i=0; i<numGPU; i++){
    hipSetDevice(i);
    hipMalloc((void**)&gpuCount[i], sizeof(unsigned) * n);
    hipMalloc((void**)&gpuFunc[i],  sizeof(float) * n * m);
    hipMalloc((void**)&gpuF1[i],    sizeof(unsigned) * chunk);
    hipMalloc((void**)&gpuF2[i],    sizeof(unsigned) * chunk);
    size = (i == numGPU - 1) ? (numPairs - i * chunk) : chunk;
    hipMemcpy(gpuCount[i], count, sizeof(unsigned) * n,
	       hipMemcpyHostToDevice);
    hipMemcpy(gpuFunc[i], func, sizeof(float) * n * m,
	       hipMemcpyHostToDevice);
    hipMemcpy(gpuF1[i], &f1[i*chunk], sizeof(unsigned) * size,
	       hipMemcpyHostToDevice);
    hipMemcpy(gpuF2[i], &f2[i*chunk], sizeof(unsigned) * size,
	       hipMemcpyHostToDevice);
    kernel<<<size, numThread>>>(gpuFunc[i], gpuCount[i], n, m,
				gpuF1[i], gpuF2[i]);
  }
  for (i=0; i<numGPU; i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
    hipMemcpy(tmpCount, gpuCount[i], sizeof(unsigned) * n,
	       hipMemcpyDeviceToHost);
    for (j=0; j<n; j++)
      count[j] += tmpCount[j];
    hipFree(gpuCount[i]);
    hipFree(gpuFunc[i]);
    hipFree(gpuF1[i]);
    hipFree(gpuF2[i]);
  }
  for (i=0; i<n; i++)
    depth[i] = (double)count[i] / (n * (n - 1.0) / 2.0);

  free(count);
  free(tmpCount);
  free(func);
  free(f1);
  free(f2);
  free(gpuCount);
  free(gpuFunc);
  free(gpuF1);
  free(gpuF2);
}
